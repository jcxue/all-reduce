#include "hip/hip_runtime.h"
#include "cuda_func.h"

__global__ void init_kernel(int64_t n, float *x, float val)
{
    int64_t index = blockIdx.x * blockDim.x + threadIdx.x;
    int64_t stride = blockDim.x * gridDim.x;
    for (int64_t i = index; i < n; i += stride) {
    	x[i] = val;
    }
}

__global__ void reduce_kernel(int64_t n, float *x, float *y)
{
    int64_t index = blockIdx.x * blockDim.x + threadIdx.x;
    int64_t stride = blockDim.x * gridDim.x;
    for (int64_t i = index; i < n; i += stride) {
    	y[i] = x[i] + y[i];
    }
}

void gpu_mem_alloc(float **x, int64_t n)
{
    hipMallocManaged(x, n*sizeof(float));
    hipDeviceSynchronize();
}

void gpu_init(int64_t n, float *x, float val)
{
    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    init_kernel<<<numBlocks, blockSize>>>(n, x, val);
    hipDeviceSynchronize();
}

void gpu_reduce(int64_t n, float *x, float *y)
{
    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    reduce_kernel<<<numBlocks, blockSize>>>(n, x, y);
    hipDeviceSynchronize();
}